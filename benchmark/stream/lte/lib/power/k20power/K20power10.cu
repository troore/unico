/*
   K20Power v1.0: This code profiles the power sensor of K20 and K40 GPUs, corrects
   the power profile, and outputs the original and corrected profiles as well as
   the energy used.  See http://cs.txstate.edu/~mb92/papers/gpgpu14.pdf for more
   detail.

   Copyright (c) 2014, Texas State University. All rights reserved.

   Redistribution and use in source and binary forms, with or without modification,
   are permitted for academic, research, experimental, or personal use provided
   that the following conditions are met:

 * Redistributions of source code must retain the above copyright notice, 
 this list of conditions and the following disclaimer.
 * Redistributions in binary form must reproduce the above copyright notice,
 this list of conditions and the following disclaimer in the documentation
 and/or other materials provided with the distribution.
 * Neither the name of Texas State University-San Marcos nor the names of its
 contributors may be used to endorse or promote products derived from this
 software without specific prior written permission.

 For all other uses, please contact the Office for Commercialization and Industry
 Relations at Texas State University-San Marcos <http://www.txstate.edu/ocir/>.

 THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS" AND
 ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
 WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
 DISCLAIMED IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR
 ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES
 (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES;
 LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON
 ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
 SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

Author: Martin Burtscher <burtscher@txstate.edu>
 */



#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>
#include <limits.h>
#include <sys/time.h>
#include <sys/types.h>
#include "nvml.h"

#define DEVICE 0  /* may need to be changed */

#define NEAR_IDLE_DELTA 500  /* mW */
#define IDLE_DELTA 250  /* mW */
#define SAMPLE_DELAY 14000  /* usec */
#define RAMP_DELAY 4000000  /* usec */
#define TIME_OUT 30000000  /* usec */
#define STABLE_COUNT 5  /* sec */

#define power2watts 0.001  /* mW -> W */
#define time2seconds 0.000001  /* usec -> sec */
#define capacitance 840000.0  /* usec */
#define ACTIVE_IDLE 55  /* W */
#define SAMPLES (1024*1024)  /* 4.3 hours */

static int samples = 0;
static int p_sample[SAMPLES];  /* power */
static long long t_sample[SAMPLES];  /* time */
static double truepower[SAMPLES];  /* true power */
static double max_power;  /* power cap in W */


static nvmlDevice_t initAndTest()
{
	nvmlReturn_t result;
	nvmlDevice_t device;
	int power;

	result = nvmlInit();
	if (NVML_SUCCESS != result) {
		printf("failed to initialize NVML: %s\n", nvmlErrorString(result));
		exit(-1);
	}

	result = nvmlDeviceGetHandleByIndex(DEVICE, &device);
	if (NVML_SUCCESS != result) {
		printf("failed to get handle for device: %s\n", nvmlErrorString(result));
		exit(-1);
	}

	result = nvmlDeviceGetPowerUsage(device, (unsigned int *)&power);
	if (NVML_SUCCESS != result) {
		printf("failed to read power: %s\n", nvmlErrorString(result));
		exit(-1);
	}

	result = nvmlDeviceGetPowerManagementLimit(device, (unsigned int *)&power);
	if (NVML_SUCCESS != result) {
		printf("failed to read power limit: %s\n", nvmlErrorString(result));
		exit(-1);
	}
	max_power = power * power2watts;

	return device;
}


static inline long long getTime()  /* usec */
{
	struct timeval time;
	gettimeofday(&time, NULL);
	return time.tv_sec * 1000000 + time.tv_usec;
}


static void getSample(nvmlDevice_t device, int *power, long long *time)  /* mW usec */
{
	nvmlReturn_t result;
	int samplepower;
	static long long sampletime = LONG_LONG_MIN;

	sampletime += SAMPLE_DELAY;
	do {} while (getTime() < sampletime);
	result = nvmlDeviceGetPowerUsage(device, (unsigned int *)&samplepower);
	sampletime = getTime();

	if (NVML_SUCCESS != result) {
		printf("failed to read power: %s\n", nvmlErrorString(result));
		exit(-1);
	}

	p_sample[samples] = samplepower;
	t_sample[samples] = sampletime;
	samples++;
	if (samples >= SAMPLES) {
		printf("out of memory for storing samples\n");
		exit(-1);
	}

	if (samples >= 3) {
		int s = samples - 2;
		double tp = (p_sample[s] + capacitance * (p_sample[s + 1] - p_sample[s - 1]) / (t_sample[s + 1] - t_sample[s - 1])) * power2watts;
		if (tp < 0.0) tp = 0.0;
		if (tp > max_power) tp = max_power;
		truepower[s] = tp;
	}

	*power = samplepower;
	*time = sampletime;
}


int main(int argc, char *argv[])
{
	int i, count, active_samples;
	nvmlDevice_t device;
	int power, prevpower, nearidlepower, diff;
	long long time, timeout, endtime;
	double activetime, activeenergy, mindt;
	FILE *f;
	char hostname[1024];
	char filename[1100];

	printf("K20Power 1.0\n");

	if (argc < 2) {
		printf("usage: %s command_line\n", argv[0]);
		exit(-1);
	}

	hostname[0] = 0;
	gethostname(hostname, 1023);
	hostname[1023] = 0;

	sprintf(filename, "K20Power_%s_%ld.trace", hostname, getTime());
	f = fopen(filename, "wt");
	fprintf(f, "K20Power 1.0\t#version\n");
	fprintf(f, "%s\t#hostname\n", hostname);
	for (i = 1; i < argc; i++) {
		fprintf(f, "%s ", argv[i]);
	}
	fprintf(f, "\t#command line\n\n");

	device = initAndTest();

	getSample(device, &power, &time);
	timeout = time + TIME_OUT;
	count = 0;
	do {
		prevpower = power;
		sleep(1);
		getSample(device, &power, &time);
		count++;
		diff = power - prevpower;
		if (diff < 0) diff = -diff;
		if (diff >= IDLE_DELTA) count = 0;
	} while ((count < STABLE_COUNT) && (time < timeout));

	if (time >= timeout) {
		printf("timed out waiting for idle power to stabilize\n");
		exit(-1);
	}

	samples = 0;
	getSample(device, &power, &time);
	endtime = time + RAMP_DELAY;
	do {
		getSample(device, &power, &time);
	} while (time < endtime);
	nearidlepower = power + NEAR_IDLE_DELTA;

	pid_t res = fork();
	if (res < 0) {
		printf("could not fork child\n");
		exit(-1);
	}
	if (res == 0) {
		execvp(argv[1], &argv[1]);
		printf("execvp returned unexpectedly");
		exit(-1);
	}

	getSample(device, &power, &time);
	timeout = time + TIME_OUT;
	do {
		getSample(device, &power, &time);
		if (power > nearidlepower) {
			timeout = time + TIME_OUT;
		}
	} while (time < timeout);

	getSample(device, &power, &time);
	getSample(device, &power, &time);

	samples--;
	active_samples = 0;
	activetime = 0.0;
	activeenergy = 0.0;
	mindt = TIME_OUT;
	for (i = 1; i < samples; i++) {
		if (truepower[i] > ACTIVE_IDLE) {
			active_samples++;
			double dt = (t_sample[i] - t_sample[i - 1]) * time2seconds;
			if (mindt > dt) mindt = dt;
			activetime += dt;
			activeenergy += dt * truepower[i];
		}
	}

	fprintf(f, "%.4f\t#active time [s]\n", activetime);
	fprintf(f, "%.4f\t#active energy [J]\n", activeenergy);
	if ((activetime - 5e-7) > 0.0) 
	{
		fprintf(f, "%.4f\t#power [W]\n", activeenergy / activetime);
	}
	else
	{
		fprintf(f, "No Energy, No Power.\n");
	}

	fprintf(f, "\ntime [s]\tpower [W]\ttrue power [W]\n");
	for (i = 1; i < samples; i++) {
		fprintf(f, "%.6f\t%.3f\t%.3f\n", (t_sample[i] - t_sample[1]) * time2seconds, p_sample[i] * power2watts, truepower[i]);
	}
	fclose(f);

	nvmlShutdown();
	return 0;
}
